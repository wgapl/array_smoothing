#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "compare.h"
#include "gputimer.h"

// Reference
__global__ void smooth(float * v_new, const float * v) {
    int myIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
    int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    //printf("%d %f %f\n", myIdx, v[myIdx], v_new[myIdx]);
}

// Your code
__global__ void smooth_shared(float * v_new, const float * v) {
    extern __shared__ float s[];
    // TODO: Complete the rest of this function
    int myId = threadIdx.x + blockIdx.x*blockDim.x;
    s[myId] = v[myId];
    __syncthreads();
    if (myId == 0) {
        v_new[myId] = 0.25f * s[myId] + 0.5f * s[myId] + 0.25f * s[myId+1];
        return;
    }
    if (myId == gridDim.x * blockDim.x - 1) {
        v_new[myId] = 0.25f * s[myId-1] + 0.5f * s[myId] + 0.25f * s[myId];
        return;
    }
    if (((myId + 1) % blockDim.x) == 0) {
        v_new[myId] = 0.25f * v[myId-1] + 0.5f * v[myId] + 0.25f * v[myId+1];
        return;
    }
    if ( (myId % blockDim.x) == 0) {
        v_new[myId] = 0.25f * v[myId-1] + 0.5f * v[myId] + 0.25f * v[myId+1];
        return;
    }
    
    v_new[myId] = 0.25f * s[myId-1] + 0.5f * s[myId] + 0.25f * s[myId+1];
    //printf("%d %f %f  %f\n", myId, s[myId], v[myId], v_new[myId]);
}

int main(int argc, char **argv)
{

    const int ARRAY_SIZE = 4096;
    const int BLOCK_SIZE = 256;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float h_cmp[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
    float h_out_shared[ARRAY_SIZE];
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [0, 1]
        h_in[i] = (float)random()/(float)RAND_MAX;
    }
    for(int i = 0; i < ARRAY_SIZE; i++) {
        h_cmp[i] = (0.25f * h_in[(i == 0) ? 0 : i-1] +
                    0.50f * h_in[i] +
                    0.25f * h_in[(i == (ARRAY_SIZE - 1)) ? ARRAY_SIZE - 1 : i+1]);
    }

    // declare GPU memory pointers
    float * d_in, * d_out, * d_out_shared;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);
    hipMalloc((void **) &d_out_shared, ARRAY_BYTES);

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // launch the kernel
    smooth<<<ARRAY_SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(d_out, d_in);
    GpuTimer timer;
    timer.Start();
    smooth_shared<<<ARRAY_SIZE / BLOCK_SIZE, BLOCK_SIZE, ARRAY_SIZE * sizeof(float)>>>(d_out_shared, d_in);
    timer.Stop();

    printf("Your code executed in %g ms\n", timer.Elapsed());
    // hipEventSynchronize(stop);
    // float elapsedTime;
    // hipEventElapsedTime(&elapsedTime, start, stop);    

    // copy back the result from GPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(h_out_shared, d_out_shared, ARRAY_BYTES, hipMemcpyDeviceToHost);
    /*for (int k = 0; k < ARRAY_SIZE; ++k)
    {
        //std::cout << h_out[k] << " " << h_cmp[k] << std::endl;
        printf("%d %f %f\n", k, h_out[k], h_cmp[k]);
    }
    */
    // testing for correctness
    compare(h_in, h_out, h_out_shared, h_cmp, ARRAY_SIZE);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shared);
}
